// vim:foldenable:foldmethod=marker:foldmarker=[[,]]
/**
 * @version 0.1.3 (2011)
 * @author Johannes Gilger <heipei@hackvalue.de>
 * 
 * Copyright 2011 Johannes Gilger
 *
 * This file is part of engine-cuda
 *
 * engine-cuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License or
 * any later version.
 * 
 * engine-cuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with engine-cuda. If not, see <http://www.gnu.org/licenses/>.
 *
 */
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#include "cuda_common.h"
#include "common.h"

#ifndef PAGEABLE
extern "C" void transferHostToDevice_PINNED   (const unsigned char *input, uint32_t *deviceMem, uint8_t *hostMem, size_t size) {
	hipError_t cudaerrno;
	if(size <= 1048576) {
		memcpy(hostMem,input,size);
        	_CUDA(hipMemcpyAsync(deviceMem, hostMem, size, hipMemcpyHostToDevice, 0));
	} else {
		//fprintf(stdout, "Now trying hipMemcpy\n");
		_CUDA(hipMemcpyAsync(deviceMem, input, size, hipMemcpyHostToDevice,0));
	}
}
#if CUDART_VERSION >= 2020
extern "C" void transferHostToDevice_ZEROCOPY (const unsigned char *input, uint32_t *deviceMem, uint8_t *hostMem, size_t size) {
	//hipError_t cudaerrno;
	memcpy(hostMem,input,size);
	//_CUDA(hipHostGetDevicePointer(&d_s,h_s, 0));
}
#endif
#else
extern "C" void transferHostToDevice_PAGEABLE (const unsigned char *input, uint32_t *deviceMem, uint8_t *hostMem, size_t size) {
	hipError_t cudaerrno;
	_CUDA(hipMemcpy(deviceMem, input, size, hipMemcpyHostToDevice));
}
#endif

#ifndef PAGEABLE
extern "C" void transferDeviceToHost_PINNED   (unsigned char *output, uint32_t *deviceMem, uint8_t *hostMemS, uint8_t *hostMemOUT, size_t size) {
	hipError_t cudaerrno;
	if(size <= 1048576) {
        	_CUDA(hipMemcpyAsync(hostMemS, deviceMem, size, hipMemcpyDeviceToHost, 0));
		_CUDA(hipDeviceSynchronize());
		memcpy(output,hostMemS,size);
	} else {
		_CUDA(hipMemcpyAsync(output, deviceMem, size, hipMemcpyDeviceToHost, 0));
	}
}
#if CUDART_VERSION >= 2020
extern "C" void transferDeviceToHost_ZEROCOPY (unsigned char *output, uint32_t *deviceMem, uint8_t *hostMemS, uint8_t *hostMemOUT, size_t size) {
	hipError_t cudaerrno;
	_CUDA(hipDeviceSynchronize());
	memcpy(output,hostMemOUT,size);
}
#endif
#else
extern "C" void transferDeviceToHost_PAGEABLE (unsigned char *output, uint32_t *deviceMem, uint8_t *hostMemS, uint8_t *hostMemOUT, size_t size) {
	hipError_t cudaerrno;
	_CUDA(hipMemcpy(output,deviceMem,size, hipMemcpyDeviceToHost));
}
#endif


float time_elapsed;
hipEvent_t time_start,time_stop;

#ifdef DEBUG
#include <sys/time.h>
int timeval_subtract (struct timeval *result, struct timeval *x, struct timeval *y) {
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (x->tv_usec - y->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  result->tv_sec = x->tv_sec - y->tv_sec;
  result->tv_usec = x->tv_usec - y->tv_usec;

  return x->tv_sec < y->tv_sec;
}
#endif

void checkCUDADevice(struct hipDeviceProp_t *deviceProp, int output_verbosity) {
	int deviceCount;
	hipError_t cudaerrno;

	_CUDA(hipGetDeviceCount(&deviceCount));

	if (!deviceCount) {
		if (output_verbosity!=OUTPUT_QUIET) 
			fprintf(stderr,"There is no device supporting CUDA.\n");
		exit(EXIT_FAILURE);
	}

	if (output_verbosity>=OUTPUT_NORMAL) 
		fprintf(stdout,"Successfully found %d CUDA devices (CUDART_VERSION %d).\n",deviceCount, CUDART_VERSION);

	/* added by abu naser(an16e@my.fsu.edu) to solve deviceid problem */	
	int deviceId;
	_CUDA(hipGetDevice(&deviceId)) ;
	if(!deviceId)
		fprintf(stdout,"Successfully found  CUDA deviceId = %d.\n",deviceId);
	else
		fprintf(stdout,"CUDA deviceId not found.\n");
	
	_CUDA(hipSetDevice(deviceId));
	_CUDA(hipGetDeviceProperties(deviceProp, deviceId));
	
	// _CUDA(hipSetDevice(6));
	// _CUDA(hipGetDeviceProperties(deviceProp, 6));
	/* end of add */
	

	
	if (output_verbosity==OUTPUT_VERBOSE) {
			fprintf(stdout,"\nDevice %d: \"%s\"\n", deviceId, deviceProp->name);
			// fprintf(stdout,"\nDevice %d: \"%s\"\n", 6, deviceProp->name);
      	 	fprintf(stdout,"  CUDA Compute Capability:                       %d.%d\n", deviceProp->major,deviceProp->minor);
#if CUDART_VERSION >= 2000
        	fprintf(stdout,"  Number of multiprocessors (SM):                %d\n", deviceProp->multiProcessorCount);
#endif
#if CUDART_VERSION >= 2020
		fprintf(stdout,"  Integrated:                                    %s\n", deviceProp->integrated ? "Yes" : "No");
        	fprintf(stdout,"  Support host page-locked memory mapping:       %s\n", deviceProp->canMapHostMemory ? "Yes" : "No");
#endif
		fprintf(stdout,"\n");
	}
}

extern "C" void cuda_device_init(int *nm, int buffer_size, int output_verbosity, uint8_t **host_data, uint64_t **device_data, uint64_t **device_data_out) {
	assert(nm);
	hipError_t cudaerrno;
	hipDeviceProp_t deviceProp;
    	
	checkCUDADevice(&deviceProp, output_verbosity);
	
	if(buffer_size==0)
		buffer_size=MAX_CHUNK_SIZE;
	
	//_CUDA(hipSetDeviceFlags(hipDeviceScheduleYield));
	//_CUDA(hipSetDeviceFlags(hipDeviceScheduleSpin));
	//_CUDA(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
	//_CUDA(hipSetDeviceFlags(hipDeviceScheduleYield|hipDeviceScheduleBlockingSync));
#if CUDART_VERSION >= 2000
	*nm=deviceProp.multiProcessorCount;
#endif

#ifndef PAGEABLE 
#if CUDART_VERSION >= 2020
	isIntegrated=deviceProp.integrated;
	if(isIntegrated) {
        	//zero-copy memory mode - use special function to get OS-pinned memory
		_CUDA(hipSetDeviceFlags(hipDeviceMapHost));
        	if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using zero-copy memory.\n");
        	_CUDA(hipHostAlloc((void**)host_data,buffer_size,hipHostMallocMapped));
		transferHostToDevice = transferHostToDevice_ZEROCOPY;		// set memory transfer function
		transferDeviceToHost = transferDeviceToHost_ZEROCOPY;		// set memory transfer function
		_CUDA(hipHostGetDevicePointer(device_data,host_data, 0));
	} else {
		//pinned memory mode - use special function to get OS-pinned memory
		_CUDA(hipHostAlloc( (void**)host_data, buffer_size, hipHostMallocDefault));
		if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pinned memory: hipHostMallocDefault.\n");
		transferHostToDevice = transferHostToDevice_PINNED;	// set memory transfer function
		transferDeviceToHost = transferDeviceToHost_PINNED;	// set memory transfer function
		_CUDA(hipMalloc((void **)device_data,buffer_size));
		_CUDA(hipMalloc((void **)device_data_out,buffer_size));
	}
#else
        //pinned memory mode - use special function to get OS-pinned memory
        _CUDA(hipHostMalloc((void**)&h_s, buffer_size));
        if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pinned memory: hipHostMallocDefault.\n");
	transferHostToDevice = transferHostToDevice_PINNED;			// set memory transfer function
	transferDeviceToHost = transferDeviceToHost_PINNED;			// set memory transfer function
	_CUDA(hipMalloc((void **)device_data,buffer_size));
	_CUDA(hipMalloc((void **)device_data_out,buffer_size));
#endif
#else
        if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pageable memory.\n");
	transferHostToDevice = transferHostToDevice_PAGEABLE;			// set memory transfer function
	transferDeviceToHost = transferDeviceToHost_PAGEABLE;			// set memory transfer function
	_CUDA(hipMalloc((void **)device_data,buffer_size));
	_CUDA(hipMalloc((void **)device_data_out,buffer_size));
#endif

	if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"The current buffer size is %d.\n\n", buffer_size);

	if(output_verbosity>=OUTPUT_NORMAL) {
		_CUDA(hipEventCreate(&time_start));
		_CUDA(hipEventCreate(&time_stop));
		_CUDA(hipEventRecord(time_start,0));
	}

}

extern "C" void cuda_device_finish(uint8_t *host_data, uint64_t *device_data) {
	hipError_t cudaerrno;

	if (output_verbosity>=OUTPUT_NORMAL) fprintf(stdout, "\nDone. Finishing up...\n");

#ifndef PAGEABLE 
#if CUDART_VERSION >= 2020
	if(isIntegrated) {
		_CUDA(hipHostFree(host_data));
	} else {
		_CUDA(hipFree(device_data));
	}
#else	
	_CUDA(hipFree(device_data));
#endif
#else
	_CUDA(hipFree(device_data));
#endif	

	if(output_verbosity>=OUTPUT_NORMAL) {
		_CUDA(hipEventRecord(time_stop,0));
		_CUDA(hipEventSynchronize(time_stop));
		_CUDA(hipEventElapsedTime(&time_elapsed,time_start,time_stop));
		fprintf(stdout,"\nTotal time: %f milliseconds\n",time_elapsed);	
	}
}
